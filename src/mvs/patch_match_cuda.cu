#include "hip/hip_runtime.h"
// Copyright (c) 2018, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#define _USE_MATH_DEFINES

#include "mvs/patch_match_cuda.h"

#include <algorithm>
#include <cfloat>
#include <cmath>
#include <cstdint>
#include <sstream>

#include "util/hip/hip_runtime.h"
#include "util/cudacc.h"
#include "util/logging.h"

// The number of threads per Cuda thread. Warning: Do not change this value,
// since the templated window sizes rely on this value.
#define THREADS_PER_BLOCK 32

// We must not include "util/math.h" to avoid any Eigen includes here,
// since Visual Studio cannot compile some of the Eigen/Boost expressions.
#ifndef DEG2RAD
#define DEG2RAD(deg) deg * 0.0174532925199432
#endif

namespace colmap {
namespace mvs {

texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat>
    ref_image_texture;
texture<uint8_t, hipTextureType2DLayered, hipReadModeNormalizedFloat>
    src_images_texture;
texture<float, hipTextureType2DLayered, hipReadModeElementType>
    src_depth_maps_texture;
texture<float, hipTextureType2D, hipReadModeElementType> poses_texture;

// Calibration of reference image as {fx, cx, fy, cy}.
__constant__ float ref_K[4];
// Calibration of reference image as {1/fx, -cx/fx, 1/fy, -cy/fy}.
__constant__ float ref_inv_K[4];

__device__ inline void Mat33DotVec3(const float mat[9], const float vec[3],
                                    float result[3]) {
  result[0] = mat[0] * vec[0] + mat[1] * vec[1] + mat[2] * vec[2];
  result[1] = mat[3] * vec[0] + mat[4] * vec[1] + mat[5] * vec[2];
  result[2] = mat[6] * vec[0] + mat[7] * vec[1] + mat[8] * vec[2];
}

__device__ inline void Mat33DotVec3Homogeneous(const float mat[9],
                                               const float vec[2],
                                               float result[2]) {
  const float inv_z = 1.0f / (mat[6] * vec[0] + mat[7] * vec[1] + mat[8]);
  result[0] = inv_z * (mat[0] * vec[0] + mat[1] * vec[1] + mat[2]);
  result[1] = inv_z * (mat[3] * vec[0] + mat[4] * vec[1] + mat[5]);
}

__device__ inline float DotProduct3(const float vec1[3], const float vec2[3]) {
  return vec1[0] * vec2[0] + vec1[1] * vec2[1] + vec1[2] * vec2[2];
}

__device__ inline float GenerateRandomDepth(const float depth_min,
                                            const float depth_max,
                                            hiprandState* rand_state) {
  return hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
}

__device__ inline void GenerateRandomNormal(const int row, const int col,
                                            hiprandState* rand_state,
                                            float normal[3]) {
  // Unbiased sampling of normal, according to George Marsaglia, "Choosing a
  // Point from the Surface of a Sphere", 1972.
  float v1 = 0.0f;
  float v2 = 0.0f;
  float s = 2.0f;
  while (s >= 1.0f) {
    v1 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    v2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    s = v1 * v1 + v2 * v2;
  }

  const float s_norm = sqrt(1.0f - s);
  normal[0] = 2.0f * v1 * s_norm;
  normal[1] = 2.0f * v2 * s_norm;
  normal[2] = 1.0f - 2.0f * s;

  // Make sure normal is looking away from camera.
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1],
                             ref_inv_K[2] * row + ref_inv_K[3], 1.0f};
  if (DotProduct3(normal, view_ray) > 0) {
    normal[0] = -normal[0];
    normal[1] = -normal[1];
    normal[2] = -normal[2];
  }
}

__device__ inline float PerturbDepth(const float perturbation,
                                     const float depth,
                                     hiprandState* rand_state) {
  const float depth_min = (1.0f - perturbation) * depth;
  const float depth_max = (1.0f + perturbation) * depth;
  return GenerateRandomDepth(depth_min, depth_max, rand_state);
}

__device__ inline void PerturbNormal(const int row, const int col,
                                     const float perturbation,
                                     const float normal[3],
                                     hiprandState* rand_state,
                                     float perturbed_normal[3],
                                     const int num_trials = 0) {
  // Perturbation rotation angles.
  const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
  const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
  const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

  const float sin_a1 = sin(a1);
  const float sin_a2 = sin(a2);
  const float sin_a3 = sin(a3);
  const float cos_a1 = cos(a1);
  const float cos_a2 = cos(a2);
  const float cos_a3 = cos(a3);

  // R = Rx * Ry * Rz
  float R[9];
  R[0] = cos_a2 * cos_a3;
  R[1] = -cos_a2 * sin_a3;
  R[2] = sin_a2;
  R[3] = cos_a1 * sin_a3 + cos_a3 * sin_a1 * sin_a2;
  R[4] = cos_a1 * cos_a3 - sin_a1 * sin_a2 * sin_a3;
  R[5] = -cos_a2 * sin_a1;
  R[6] = sin_a1 * sin_a3 - cos_a1 * cos_a3 * sin_a2;
  R[7] = cos_a3 * sin_a1 + cos_a1 * sin_a2 * sin_a3;
  R[8] = cos_a1 * cos_a2;

  // Perturb the normal vector.
  Mat33DotVec3(R, normal, perturbed_normal);

  // Make sure the perturbed normal is still looking in the same direction as
  // the viewing direction, otherwise try again but with smaller perturbation.
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1],
                             ref_inv_K[2] * row + ref_inv_K[3], 1.0f};
  if (DotProduct3(perturbed_normal, view_ray) >= 0.0f) {
    const int kMaxNumTrials = 3;
    if (num_trials < kMaxNumTrials) {
      PerturbNormal(row, col, 0.5f * perturbation, normal, rand_state,
                    perturbed_normal, num_trials + 1);
      return;
    } else {
      perturbed_normal[0] = normal[0];
      perturbed_normal[1] = normal[1];
      perturbed_normal[2] = normal[2];
      return;
    }
  }

  // Make sure normal has unit norm.
  const float inv_norm = rsqrt(DotProduct3(perturbed_normal, perturbed_normal));
  perturbed_normal[0] *= inv_norm;
  perturbed_normal[1] *= inv_norm;
  perturbed_normal[2] *= inv_norm;
}

__device__ inline void ComputePointAtDepth(const float row, const float col,
                                           const float depth, float point[3]) {
  point[0] = depth * (ref_inv_K[0] * col + ref_inv_K[1]);
  point[1] = depth * (ref_inv_K[2] * row + ref_inv_K[3]);
  point[2] = depth;
}

// Transfer depth on plane from viewing ray at row1 to row2. The returned
// depth is the intersection of the viewing ray through row2 with the plane
// at row1 defined by the given depth and normal.
__device__ inline float PropagateDepth(const float depth1,
                                       const float normal1[3], const float row1,
                                       const float row2) {
  // Point along first viewing ray.
  const float x1 = depth1 * (ref_inv_K[2] * row1 + ref_inv_K[3]);
  const float y1 = depth1;
  // Point on plane defined by point along first viewing ray and plane normal1.
  const float x2 = x1 + normal1[2];
  const float y2 = y1 - normal1[1];

  // Origin of second viewing ray.
  // const float x3 = 0.0f;
  // const float y3 = 0.0f;
  // Point on second viewing ray.
  const float x4 = ref_inv_K[2] * row2 + ref_inv_K[3];
  // const float y4 = 1.0f;

  // Intersection of the lines ((x1, y1), (x2, y2)) and ((x3, y3), (x4, y4)).
  const float denom = x2 - x1 + x4 * (y1 - y2);
  constexpr float kEps = 1e-5f;
  if (abs(denom) < kEps) {
    return depth1;
  }
  const float nom = y1 * x2 - x1 * y2;
  return nom / denom;
}

// First, compute triangulation angle between reference and source image for 3D
// point. Second, compute incident angle between viewing direction of source
// image and normal direction of 3D point. Both angles are cosine distances.
__device__ inline void ComputeViewingAngles(const float point[3],
                                            const float normal[3],
                                            const int image_idx,
                                            float* cos_triangulation_angle,
                                            float* cos_incident_angle) {
  *cos_triangulation_angle = 0.0f;
  *cos_incident_angle = 0.0f;

  // Projection center of source image.
  float C[3];
  for (int i = 0; i < 3; ++i) {
    C[i] = tex2D(poses_texture, i + 16, image_idx);
  }

  // Ray from point to camera.
  const float SX[3] = {C[0] - point[0], C[1] - point[1], C[2] - point[2]};

  // Length of ray from reference image to point.
  const float RX_inv_norm = rsqrt(DotProduct3(point, point));

  // Length of ray from source image to point.
  const float SX_inv_norm = rsqrt(DotProduct3(SX, SX));

  *cos_incident_angle = DotProduct3(SX, normal) * SX_inv_norm;
  *cos_triangulation_angle = DotProduct3(SX, point) * RX_inv_norm * SX_inv_norm;
}

__device__ inline void ComposeHomography(const int image_idx, const int row,
                                         const int col, const float depth,
                                         const float normal[3], float H[9]) {
  // Calibration of source image.
  float K[4];
  for (int i = 0; i < 4; ++i) {
    K[i] = tex2D(poses_texture, i, image_idx);
  }

  // Relative rotation between reference and source image.
  float R[9];
  for (int i = 0; i < 9; ++i) {
    R[i] = tex2D(poses_texture, i + 4, image_idx);
  }

  // Relative translation between reference and source image.
  float T[3];
  for (int i = 0; i < 3; ++i) {
    T[i] = tex2D(poses_texture, i + 13, image_idx);
  }

  // Distance to the plane.
  const float dist =
      depth * (normal[0] * (ref_inv_K[0] * col + ref_inv_K[1]) +
               normal[1] * (ref_inv_K[2] * row + ref_inv_K[3]) + normal[2]);
  const float inv_dist = 1.0f / dist;

  const float inv_dist_N0 = inv_dist * normal[0];
  const float inv_dist_N1 = inv_dist * normal[1];
  const float inv_dist_N2 = inv_dist * normal[2];

  // Homography as H = K * (R - T * n' / d) * Kref^-1.
  H[0] = ref_inv_K[0] * (K[0] * (R[0] + inv_dist_N0 * T[0]) +
                         K[1] * (R[6] + inv_dist_N0 * T[2]));
  H[1] = ref_inv_K[2] * (K[0] * (R[1] + inv_dist_N1 * T[0]) +
                         K[1] * (R[7] + inv_dist_N1 * T[2]));
  H[2] = K[0] * (R[2] + inv_dist_N2 * T[0]) +
         K[1] * (R[8] + inv_dist_N2 * T[2]) +
         ref_inv_K[1] * (K[0] * (R[0] + inv_dist_N0 * T[0]) +
                         K[1] * (R[6] + inv_dist_N0 * T[2])) +
         ref_inv_K[3] * (K[0] * (R[1] + inv_dist_N1 * T[0]) +
                         K[1] * (R[7] + inv_dist_N1 * T[2]));
  H[3] = ref_inv_K[0] * (K[2] * (R[3] + inv_dist_N0 * T[1]) +
                         K[3] * (R[6] + inv_dist_N0 * T[2]));
  H[4] = ref_inv_K[2] * (K[2] * (R[4] + inv_dist_N1 * T[1]) +
                         K[3] * (R[7] + inv_dist_N1 * T[2]));
  H[5] = K[2] * (R[5] + inv_dist_N2 * T[1]) +
         K[3] * (R[8] + inv_dist_N2 * T[2]) +
         ref_inv_K[1] * (K[2] * (R[3] + inv_dist_N0 * T[1]) +
                         K[3] * (R[6] + inv_dist_N0 * T[2])) +
         ref_inv_K[3] * (K[2] * (R[4] + inv_dist_N1 * T[1]) +
                         K[3] * (R[7] + inv_dist_N1 * T[2]));
  H[6] = ref_inv_K[0] * (R[6] + inv_dist_N0 * T[2]);
  H[7] = ref_inv_K[2] * (R[7] + inv_dist_N1 * T[2]);
  H[8] = R[8] + ref_inv_K[1] * (R[6] + inv_dist_N0 * T[2]) +
         ref_inv_K[3] * (R[7] + inv_dist_N1 * T[2]) + inv_dist_N2 * T[2];
}

// Each thread in the current warp / thread block reads in 3 columns of the
// reference image. The shared memory holds 3 * THREADS_PER_BLOCK columns and
// kWindowSize rows of the reference image. Each thread copies every
// THREADS_PER_BLOCK-th column from global to shared memory offset by its ID.
// For example, if THREADS_PER_BLOCK = 32, then thread 0 reads columns 0, 32, 64
// and thread 1 columns 1, 33, 65. When computing the photoconsistency, which is
// shared among each thread block, each thread can then read the reference image
// colors from shared memory. Note that this limits the window radius to a
// maximum of THREADS_PER_BLOCK.
template <int kWindowSize>
struct LocalRefImage {
  const static int kWindowRadius = kWindowSize / 2;
  const static int kThreadBlockRadius = 1;
  const static int kThreadBlockSize = 2 * kThreadBlockRadius + 1;
  const static int kNumRows = kWindowSize;
  const static int kNumColumns = kThreadBlockSize * THREADS_PER_BLOCK;
  const static int kDataSize = kNumRows * kNumColumns;

  float* data = nullptr;

  __device__ inline void Read(const int row) {
    // For the first row, read the entire block into shared memory. For all
    // consecutive rows, it is only necessary to shift the rows in shared memory
    // up by one element and then read in a new row at the bottom of the shared
    // memory. Note that this assumes that the calling loop starts with the
    // first row and then consecutively reads in the next row.

    const int thread_id = threadIdx.x;
    const int thread_block_first_id = blockDim.x * blockIdx.x;

    const int local_col_start = thread_id;
    const int global_col_start = thread_block_first_id -
                                 kThreadBlockRadius * THREADS_PER_BLOCK +
                                 thread_id;

    if (row == 0) {
      int global_row = row - kWindowRadius;
      for (int local_row = 0; local_row < kNumRows; ++local_row, ++global_row) {
        int local_col = local_col_start;
        int global_col = global_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[local_row * kNumColumns + local_col] =
              tex2D(ref_image_texture, global_col, global_row);
          local_col += THREADS_PER_BLOCK;
          global_col += THREADS_PER_BLOCK;
        }
      }
    } else {
      // Move rows in shared memory up by one row.
      for (int local_row = 1; local_row < kNumRows; ++local_row) {
        int local_col = local_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[(local_row - 1) * kNumColumns + local_col] =
              data[local_row * kNumColumns + local_col];
          local_col += THREADS_PER_BLOCK;
        }
      }

      // Read next row into the last row of shared memory.
      const int local_row = kNumRows - 1;
      const int global_row = row + kWindowRadius;
      int local_col = local_col_start;
      int global_col = global_col_start;
#pragma unroll
      for (int block = 0; block < kThreadBlockSize; ++block) {
        data[local_row * kNumColumns + local_col] =
            tex2D(ref_image_texture, global_col, global_row);
        local_col += THREADS_PER_BLOCK;
        global_col += THREADS_PER_BLOCK;
      }
    }
  }
};

// The return values is 1 - NCC, so the range is [0, 2], the smaller the
// value, the better the color consistency.
template <int kWindowSize, int kWindowStep>
struct PhotoConsistencyCostComputer {
  const static int kWindowRadius = kWindowSize / 2;

  __device__ PhotoConsistencyCostComputer(const float sigma_spatial,
                                          const float sigma_color)
      : bilateral_weight_computer_(sigma_spatial, sigma_color) {}

  // Maximum photo consistency cost as 1 - min(NCC).
  const float kMaxCost = 2.0f;

  // Thread warp local reference image data around current patch.
  typedef LocalRefImage<kWindowSize> LocalRefImageType;
  LocalRefImageType local_ref_image;

  // Precomputed sum of raw and squared image intensities.
  float local_ref_sum = 0.0f;
  float local_ref_squared_sum = 0.0f;

  // Index of source image.
  int src_image_idx = -1;

  // Center position of patch in reference image.
  int row = -1;
  int col = -1;

  // Depth and normal for which to warp patch.
  float depth = 0.0f;
  const float* normal = nullptr;

  __device__ inline void Read(const int row) {
    local_ref_image.Read(row);
    __syncthreads();
  }

  __device__ inline float Compute() const {
    float tform[9];
    ComposeHomography(src_image_idx, row, col, depth, normal, tform);

    float tform_step[8];
    for (int i = 0; i < 8; ++i) {
      tform_step[i] = kWindowStep * tform[i];
    }

    const int thread_id = threadIdx.x;
    const int row_start = row - kWindowRadius;
    const int col_start = col - kWindowRadius;

    float col_src = tform[0] * col_start + tform[1] * row_start + tform[2];
    float row_src = tform[3] * col_start + tform[4] * row_start + tform[5];
    float z = tform[6] * col_start + tform[7] * row_start + tform[8];
    float base_col_src = col_src;
    float base_row_src = row_src;
    float base_z = z;

    int ref_image_idx = THREADS_PER_BLOCK - kWindowRadius + thread_id;
    int ref_image_base_idx = ref_image_idx;

    const float ref_center_color =
        local_ref_image
            .data[ref_image_idx + kWindowRadius * 3 * THREADS_PER_BLOCK +
                  kWindowRadius];
    const float ref_color_sum = local_ref_sum;
    const float ref_color_squared_sum = local_ref_squared_sum;
    float src_color_sum = 0.0f;
    float src_color_squared_sum = 0.0f;
    float src_ref_color_sum = 0.0f;
    float bilateral_weight_sum = 0.0f;

    for (int row = -kWindowRadius; row <= kWindowRadius; row += kWindowStep) {
      for (int col = -kWindowRadius; col <= kWindowRadius; col += kWindowStep) {
        const float inv_z = 1.0f / z;
        const float norm_col_src = inv_z * col_src + 0.5f;
        const float norm_row_src = inv_z * row_src + 0.5f;
        const float ref_color = local_ref_image.data[ref_image_idx];
        const float src_color = tex2DLayered(src_images_texture, norm_col_src,
                                             norm_row_src, src_image_idx);

        const float bilateral_weight = bilateral_weight_computer_.Compute(
            row, col, ref_center_color, ref_color);

        const float bilateral_weight_src = bilateral_weight * src_color;

        src_color_sum += bilateral_weight_src;
        src_color_squared_sum += bilateral_weight_src * src_color;
        src_ref_color_sum += bilateral_weight_src * ref_color;
        bilateral_weight_sum += bilateral_weight;

        ref_image_idx += kWindowStep;

        // Accumulate warped source coordinates per row to reduce numerical
        // errors. Note that this is necessary since coordinates usually are in
        // the order of 1000s as opposed to the color values which are
        // normalized to the range [0, 1].
        col_src += tform_step[0];
        row_src += tform_step[3];
        z += tform_step[6];
      }

      ref_image_base_idx += kWindowStep * 3 * THREADS_PER_BLOCK;
      ref_image_idx = ref_image_base_idx;

      base_col_src += tform_step[1];
      base_row_src += tform_step[4];
      base_z += tform_step[7];

      col_src = base_col_src;
      row_src = base_row_src;
      z = base_z;
    }

    const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
    src_color_sum *= inv_bilateral_weight_sum;
    src_color_squared_sum *= inv_bilateral_weight_sum;
    src_ref_color_sum *= inv_bilateral_weight_sum;

    const float ref_color_var =
        ref_color_squared_sum - ref_color_sum * ref_color_sum;
    const float src_color_var =
        src_color_squared_sum - src_color_sum * src_color_sum;

    // Based on Jensen's Inequality for convex functions, the variance
    // should always be larger than 0. Do not make this threshold smaller.
    constexpr float kMinVar = 1e-5f;
    if (ref_color_var < kMinVar || src_color_var < kMinVar) {
      return kMaxCost;
    } else {
      const float src_ref_color_covar =
          src_ref_color_sum - ref_color_sum * src_color_sum;
      const float src_ref_color_var = sqrt(ref_color_var * src_color_var);
      return max(0.0f,
                 min(kMaxCost, 1.0f - src_ref_color_covar / src_ref_color_var));
    }
  }

  __device__ inline float ACMMCompute_shared(const int thread_idx_x, const int thread_idx_y) const {
  	

    float tform[9];
    ComposeHomography(src_image_idx, row, col, depth, normal, tform);

    float tform_step[9];
    for (int i = 0; i < 9; ++i) {
      tform_step[i] = kWindowStep * tform[i];
    }

    const int row_start = row - kWindowRadius;
    const int col_start = col - kWindowRadius;

    float col_src = tform[0] * col_start + tform[1] * row_start + tform[2];
    float row_src = tform[3] * col_start + tform[4] * row_start + tform[5];
    float z = tform[6] * col_start + tform[7] * row_start + tform[8];
    float base_col_src = col_src;
    float base_row_src = row_src;
    float base_z = z;

    // shared memory
 //    const int shared_width = 3 * THREADS_PER_BLOCK;
 //    const int shared_idx = THREADS_PER_BLOCK + thread_idx_x;
	// const int shared_idy = THREADS_PER_BLOCK + thread_idx_y;
	// const int cur_idx = shared_idy * shared_width + shared_idx;
 //    int ref_image_idx = cur_idx - kWindowRadius * shared_width- kWindowRadius;
 //    int ref_image_base_idx = ref_image_idx;
    const int shared_width = (THREADS_PER_BLOCK + 2 * kWindowRadius);
    const int cur_idx = (thread_idx_y + kWindowRadius) * shared_width+ kWindowRadius + thread_idx_x;

    int ref_image_idx = cur_idx - kWindowRadius * shared_width- kWindowRadius;
    int ref_image_base_idx = ref_image_idx;

    const float ref_center_color = local_ref_image[cur_idx];
    
    const float ref_color_sum = local_ref_sum;
    const float ref_color_squared_sum = local_ref_squared_sum;
    float src_color_sum = 0.0f;
    float src_color_squared_sum = 0.0f;
    float src_ref_color_sum = 0.0f;
    float bilateral_weight_sum = 0.0f;

    //int index = 0;
    for (int r = -kWindowRadius; r <= kWindowRadius; r += kWindowStep) {
      for (int c = -kWindowRadius; c <= kWindowRadius; c += kWindowStep) {
       const float inv_z = 1.0f / z;
        const float norm_col_src = inv_z * col_src + 0.5f;
        const float norm_row_src = inv_z * row_src + 0.5f;
        
        const float ref_color = local_ref_image[ref_image_idx];//tex2D(ref_image_texture, col + c, row + r);//
        const float src_color = tex2DLayered(src_images_texture, norm_col_src,
                                             norm_row_src, src_image_idx);

        const float bilateral_weight = bilateral_weight_computer_.Compute(
            r, c, ref_center_color, ref_color);

        const float bilateral_weight_src = bilateral_weight * src_color;

        src_color_sum += bilateral_weight_src;
        src_color_squared_sum += bilateral_weight_src * src_color;
        src_ref_color_sum += bilateral_weight_src * ref_color;
        bilateral_weight_sum += bilateral_weight;

        ref_image_idx += kWindowStep;

        // Accumulate warped source coordinates per row to reduce numerical
        // errors. Note that this is necessary since coordinates usually are in
        // the order of 1000s as opposed to the color values which are
        // normalized to the range [0, 1].
        col_src += tform_step[0];
        row_src += tform_step[3];
        z += tform_step[6];
      	
 
      }

      ref_image_base_idx += kWindowStep * shared_width;
      ref_image_idx = ref_image_base_idx;

      base_col_src += tform_step[1];
      base_row_src += tform_step[4];
      base_z += tform_step[7];

      col_src = base_col_src;
      row_src = base_row_src;
      z = base_z;
    }

    const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
    src_color_sum *= inv_bilateral_weight_sum;
    src_color_squared_sum *= inv_bilateral_weight_sum;
    src_ref_color_sum *= inv_bilateral_weight_sum;

    const float ref_color_var =
        ref_color_squared_sum - ref_color_sum * ref_color_sum;
    const float src_color_var =
        src_color_squared_sum - src_color_sum * src_color_sum;

    // Based on Jensen's Inequality for convex functions, the variance
    // should always be larger than 0. Do not make this threshold smaller.
    const float kMinVar = 1e-5f;
    //printf("%f\n", ref_center_color);
    if (ref_color_var < kMinVar || src_color_var < kMinVar) {
      return kMaxCost;
    } else {
      const float src_ref_color_covar =
          src_ref_color_sum - ref_color_sum * src_color_sum;
      const float src_ref_color_var = sqrt(ref_color_var * src_color_var);
      return max(0.0f,
                 min(kMaxCost, 1.0f - src_ref_color_covar / src_ref_color_var));
    }
    //return 0.7f;
  }

 private:
  const BilateralWeightComputer bilateral_weight_computer_;
};

__device__ inline float ComputeGeomConsistencyCost(const float row,
                                                   const float col,
                                                   const float depth,
                                                   const int image_idx,
                                                   const float max_cost) {
  // Extract projection matrices for source image.
  float P[12];
  for (int i = 0; i < 12; ++i) {
    P[i] = tex2D(poses_texture, i + 19, image_idx);
  }
  float inv_P[12];
  for (int i = 0; i < 12; ++i) {
    inv_P[i] = tex2D(poses_texture, i + 31, image_idx);
  }

  // Project point in reference image to world.
  float forward_point[3];
  ComputePointAtDepth(row, col, depth, forward_point);

  // Project world point to source image.
  const float inv_forward_z =
      1.0f / (P[8] * forward_point[0] + P[9] * forward_point[1] +
              P[10] * forward_point[2] + P[11]);
  float src_col =
      inv_forward_z * (P[0] * forward_point[0] + P[1] * forward_point[1] +
                       P[2] * forward_point[2] + P[3]);
  float src_row =
      inv_forward_z * (P[4] * forward_point[0] + P[5] * forward_point[1] +
                       P[6] * forward_point[2] + P[7]);

  // Extract depth in source image.
  const float src_depth = tex2DLayered(src_depth_maps_texture, src_col + 0.5f,
                                       src_row + 0.5f, image_idx);

  // Projection outside of source image.
  if (src_depth == 0.0f) {
    return max_cost;
  }

  // Project point in source image to world.
  src_col *= src_depth;
  src_row *= src_depth;
  const float backward_point_x =
      inv_P[0] * src_col + inv_P[1] * src_row + inv_P[2] * src_depth + inv_P[3];
  const float backward_point_y =
      inv_P[4] * src_col + inv_P[5] * src_row + inv_P[6] * src_depth + inv_P[7];
  const float backward_point_z = inv_P[8] * src_col + inv_P[9] * src_row +
                                 inv_P[10] * src_depth + inv_P[11];
  const float inv_backward_point_z = 1.0f / backward_point_z;

  // Project world point back to reference image.
  const float backward_col =
      inv_backward_point_z *
      (ref_K[0] * backward_point_x + ref_K[1] * backward_point_z);
  const float backward_row =
      inv_backward_point_z *
      (ref_K[2] * backward_point_y + ref_K[3] * backward_point_z);

  // Return truncated reprojection error between original observation and
  // the forward-backward projected observation.
  const float diff_col = col - backward_col;
  const float diff_row = row - backward_row;
  return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

// Find index of minimum in given values.
template <int kNumCosts>
__device__ inline int FindMinCost(const float costs[kNumCosts]) {
  float min_cost = costs[0];
  int min_cost_idx = 0;
  for (int idx = 1; idx < kNumCosts; ++idx) {
    if (costs[idx] <= min_cost) {
      min_cost = costs[idx];
      min_cost_idx = idx;
    }
  }
  return min_cost_idx;
}

__device__ inline void TransformPDFToCDF(float* probs, const int num_probs) {
  float prob_sum = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    prob_sum += probs[i];
  }
  const float inv_prob_sum = 1.0f / prob_sum;

  float cum_prob = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    const float prob = probs[i] * inv_prob_sum;
    cum_prob += prob;
    probs[i] = cum_prob;
  }
}

class LikelihoodComputer {
 public:
  __device__ LikelihoodComputer(const float ncc_sigma,
                                const float min_triangulation_angle,
                                const float incident_angle_sigma)
      : cos_min_triangulation_angle_(cos(min_triangulation_angle)),
        inv_incident_angle_sigma_square_(
            -0.5f / (incident_angle_sigma * incident_angle_sigma)),
        inv_ncc_sigma_square_(-0.5f / (ncc_sigma * ncc_sigma)),
        ncc_norm_factor_(ComputeNCCCostNormFactor(ncc_sigma)) {}

  // Compute forward message from current cost and forward message of
  // previous / neighboring pixel.
  __device__ float ComputeForwardMessage(const float cost,
                                         const float prev) const {
    return ComputeMessage<true>(cost, prev);
  }

  // Compute backward message from current cost and backward message of
  // previous / neighboring pixel.
  __device__ float ComputeBackwardMessage(const float cost,
                                          const float prev) const {
    return ComputeMessage<false>(cost, prev);
  }

  // Compute the selection probability from the forward and backward message.
  __device__ inline float ComputeSelProb(const float alpha, const float beta,
                                         const float prev,
                                         const float prev_weight) const {
    const float zn0 = (1.0f - alpha) * (1.0f - beta);
    const float zn1 = alpha * beta;
    const float curr = zn1 / (zn0 + zn1);
    return prev_weight * prev + (1.0f - prev_weight) * curr;
  }

  // Compute NCC probability. Note that cost = 1 - NCC.
  __device__ inline float ComputeNCCProb(const float cost) const {
    return exp(cost * cost * inv_ncc_sigma_square_) * ncc_norm_factor_;
  }

  // Compute the triangulation angle probability.
  __device__ inline float ComputeTriProb(
      const float cos_triangulation_angle) const {
    const float abs_cos_triangulation_angle = abs(cos_triangulation_angle);
    if (abs_cos_triangulation_angle > cos_min_triangulation_angle_) {
      const float scaled = 1.0f - (1.0f - abs_cos_triangulation_angle) /
                                      (1.0f - cos_min_triangulation_angle_);
      const float likelihood = 1.0f - scaled * scaled;
      return min(1.0f, max(0.0f, likelihood));
    } else {
      return 1.0f;
    }
  }

  // Compute the incident angle probability.
  __device__ inline float ComputeIncProb(const float cos_incident_angle) const {
    const float x = 1.0f - max(0.0f, cos_incident_angle);
    return exp(x * x * inv_incident_angle_sigma_square_);
  }

  // Compute the warping/resolution prior probability.
  template <int kWindowSize>
  __device__ inline float ComputeResolutionProb(const float H[9],
                                                const float row,
                                                const float col) const {
    const int kWindowRadius = kWindowSize / 2;

    // Warp corners of patch in reference image to source image.
    float src1[2];
    const float ref1[2] = {col - kWindowRadius, row - kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref1, src1);
    float src2[2];
    const float ref2[2] = {col - kWindowRadius, row + kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref2, src2);
    float src3[2];
    const float ref3[2] = {col + kWindowRadius, row + kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref3, src3);
    float src4[2];
    const float ref4[2] = {col + kWindowRadius, row - kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref4, src4);

    // Compute area of patches in reference and source image.
    const float ref_area = kWindowSize * kWindowSize;
    const float src_area =
        abs(0.5f * (src1[0] * src2[1] - src2[0] * src1[1] - src1[0] * src4[1] +
                    src2[0] * src3[1] - src3[0] * src2[1] + src4[0] * src1[1] +
                    src3[0] * src4[1] - src4[0] * src3[1]));

    if (ref_area > src_area) {
      return src_area / ref_area;
    } else {
      return ref_area / src_area;
    }
  }

 private:
  // The normalization for the likelihood function, i.e. the normalization for
  // the prior on the matching cost.
  __device__ static inline float ComputeNCCCostNormFactor(
      const float ncc_sigma) {
    // A = sqrt(2pi)*sigma/2*erf(sqrt(2)/sigma)
    // erf(x) = 2/sqrt(pi) * integral from 0 to x of exp(-t^2) dt
    return 2.0f / (sqrt(2.0f * M_PI) * ncc_sigma *
                   erff(2.0f / (ncc_sigma * 1.414213562f)));
  }

  // Compute the forward or backward message.
  template <bool kForward>
  __device__ inline float ComputeMessage(const float cost,
                                         const float prev) const {
    constexpr float kUniformProb = 0.5f;
    constexpr float kNoChangeProb = 0.99999f;
    const float kChangeProb = 1.0f - kNoChangeProb;
    const float emission = ComputeNCCProb(cost);

    float zn0;  // Message for selection probability = 0.
    float zn1;  // Message for selection probability = 1.
    if (kForward) {
      zn0 = (prev * kChangeProb + (1.0f - prev) * kNoChangeProb) * kUniformProb;
      zn1 = (prev * kNoChangeProb + (1.0f - prev) * kChangeProb) * emission;
    } else {
      zn0 = prev * emission * kChangeProb +
            (1.0f - prev) * kUniformProb * kNoChangeProb;
      zn1 = prev * emission * kNoChangeProb +
            (1.0f - prev) * kUniformProb * kChangeProb;
    }

    return zn1 / (zn0 + zn1);
  }

  float cos_min_triangulation_angle_;
  float inv_incident_angle_sigma_square_;
  float inv_ncc_sigma_square_;
  float ncc_norm_factor_;
};

// Rotate normals by 90deg around z-axis in counter-clockwise direction.
__global__ void InitNormalMap(GpuMat<float> normal_map,
                              GpuMat<hiprandState> rand_state_map) {
  const int row = blockDim.y * blockIdx.y + threadIdx.y;
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  if (col < normal_map.GetWidth() && row < normal_map.GetHeight()) {
    hiprandState rand_state = rand_state_map.Get(row, col);
    float normal[3];
    GenerateRandomNormal(row, col, &rand_state, normal);
    normal_map.SetSlice(row, col, normal);
    rand_state_map.Set(row, col, rand_state);
  }
}

// Rotate normals by 90deg around z-axis in counter-clockwise direction.
__global__ void RotateNormalMap(GpuMat<float> normal_map) {
  const int row = blockDim.y * blockIdx.y + threadIdx.y;
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  if (col < normal_map.GetWidth() && row < normal_map.GetHeight()) {
    float normal[3];
    normal_map.GetSlice(row, col, normal);
    float rotated_normal[3];
    rotated_normal[0] = normal[1];
    rotated_normal[1] = -normal[0];
    rotated_normal[2] = normal[2];
    normal_map.SetSlice(row, col, rotated_normal);
  }
}

template <int kWindowSize, int kWindowStep>
__global__ void ComputeInitialCost(GpuMat<float> cost_map,
                                   const GpuMat<float> depth_map,
                                   const GpuMat<float> normal_map,
                                   const GpuMat<float> ref_sum_image,
                                   const GpuMat<float> ref_squared_sum_image,
                                   const float sigma_spatial,
                                   const float sigma_color) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(sigma_spatial, sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  float normal[3] = {0};
  pcc_computer.normal = normal;

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col < cost_map.GetWidth()) {
      pcc_computer.depth = depth_map.Get(row, col);
      normal_map.GetSlice(row, col, normal);

      pcc_computer.row = row;
      pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
      pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        pcc_computer.src_image_idx = image_idx;
        cost_map.Set(row, col, image_idx, pcc_computer.Compute());
      }
    }
  }
}

struct SweepOptions {
  float perturbation = 1.0f;
  float depth_min = 0.0f;
  float depth_max = 1.0f;
  int num_samples = 15;
  float sigma_spatial = 3.0f;
  float sigma_color = 0.3f;
  float ncc_sigma = 0.6f;
  float min_triangulation_angle = 0.5f;
  float incident_angle_sigma = 0.9f;
  float prev_sel_prob_weight = 0.0f;
  float geom_consistency_regularizer = 0.1f;
  float geom_consistency_max_cost = 5.0f;
  float filter_min_ncc = 0.1f;
  float filter_min_triangulation_angle = 3.0f;
  int filter_min_num_consistent = 2;
  float filter_geom_consistency_max_cost = 1.0f;
};

template <int kWindowSize, int kWindowStep, bool kGeomConsistencyTerm = false,
          bool kFilterPhotoConsistency = false,
          bool kFilterGeomConsistency = false>
__global__ void SweepFromTopToBottom(
    GpuMat<float> global_workspace, GpuMat<hiprandState> rand_state_map,
    GpuMat<float> cost_map, GpuMat<float> depth_map, GpuMat<float> normal_map,
    GpuMat<uint8_t> consistency_mask, GpuMat<float> sel_prob_map,
    const GpuMat<float> prev_sel_prob_map, const GpuMat<float> ref_sum_image,
    const GpuMat<float> ref_squared_sum_image, const SweepOptions options) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  // Probability for boundary pixels.
  constexpr float kUniformProb = 0.5f;

  LikelihoodComputer likelihood_computer(options.ncc_sigma,
                                         options.min_triangulation_angle,
                                         options.incident_angle_sigma);

  float* forward_message =
      &global_workspace.GetPtr()[col * global_workspace.GetHeight()];
  float* sampling_probs =
      &global_workspace.GetPtr()[global_workspace.GetWidth() *
                                     global_workspace.GetHeight() +
                                 col * global_workspace.GetHeight()];

  //////////////////////////////////////////////////////////////////////////////
  // Compute backward message for all rows. Note that the backward messages are
  // temporarily stored in the sel_prob_map and replaced row by row as the
  // updated forward messages are computed further below.
  //////////////////////////////////////////////////////////////////////////////

  if (col < cost_map.GetWidth()) {
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Compute backward message.
      float beta = kUniformProb;
      for (int row = cost_map.GetHeight() - 1; row >= 0; --row) {
        const float cost = cost_map.Get(row, col, image_idx);
        beta = likelihood_computer.ComputeBackwardMessage(cost, beta);
        sel_prob_map.Set(row, col, image_idx, beta);
      }

      // Initialize forward message.
      forward_message[image_idx] = kUniformProb;
    }
  }

  //////////////////////////////////////////////////////////////////////////////
  // Estimate parameters for remaining rows and compute selection probabilities.
  //////////////////////////////////////////////////////////////////////////////

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(options.sigma_spatial,
                                                options.sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  struct ParamState {
    float depth = 0.0f;
    float normal[3] = {0};
  };

  // Parameters of previous pixel in column.
  ParamState prev_param_state;
  // Parameters of current pixel in column.
  ParamState curr_param_state;
  // Randomly sampled parameters.
  ParamState rand_param_state;
  // Cuda PRNG state for random sampling.
  hiprandState rand_state;

  if (col < cost_map.GetWidth()) {
    // Read random state for current column.
    rand_state = rand_state_map.Get(0, col);
    // Parameters for first row in column.
    prev_param_state.depth = depth_map.Get(0, col);
    normal_map.GetSlice(0, col, prev_param_state.normal);
  }

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col >= cost_map.GetWidth()) {
      continue;
    }

    pcc_computer.row = row;
    pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
    pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

    // Propagate the depth at which the current ray intersects with the plane
    // of the normal of the previous ray. This helps to better estimate
    // the depth of very oblique structures, i.e. pixels whose normal direction
    // is significantly different from their viewing direction.
    prev_param_state.depth = PropagateDepth(
        prev_param_state.depth, prev_param_state.normal, row - 1, row);

    // Read parameters for current pixel from previous sweep.
    curr_param_state.depth = depth_map.Get(row, col);
    normal_map.GetSlice(row, col, curr_param_state.normal);

    // Generate random parameters.
    rand_param_state.depth =
        PerturbDepth(options.perturbation, curr_param_state.depth, &rand_state);
    PerturbNormal(row, col, options.perturbation * M_PI,
                  curr_param_state.normal, &rand_state,
                  rand_param_state.normal);

    // Read in the backward message, compute selection probabilities and
    // modulate selection probabilities with priors.

    float point[3];
    ComputePointAtDepth(row, col, curr_param_state.depth, point);

    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      const float cost = cost_map.Get(row, col, image_idx);
      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float sel_prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);

      float cos_triangulation_angle;
      float cos_incident_angle;
      ComputeViewingAngles(point, curr_param_state.normal, image_idx,
                           &cos_triangulation_angle, &cos_incident_angle);
      const float tri_prob =
          likelihood_computer.ComputeTriProb(cos_triangulation_angle);
      const float inc_prob =
          likelihood_computer.ComputeIncProb(cos_incident_angle);

      float H[9];
      ComposeHomography(image_idx, row, col, curr_param_state.depth,
                        curr_param_state.normal, H);
      const float res_prob =
          likelihood_computer.ComputeResolutionProb<kWindowSize>(H, row, col);

      sampling_probs[image_idx] = sel_prob * tri_prob * inc_prob * res_prob;
    }

    TransformPDFToCDF(sampling_probs, cost_map.GetDepth());

    // Compute matching cost using Monte Carlo sampling of source images. Images
    // with higher selection probability are more likely to be sampled. Hence,
    // if only very few source images see the reference image pixel, the same
    // source image is likely to be sampled many times. Instead of taking
    // the best K probabilities, this sampling scheme has the advantage of
    // being adaptive to any distribution of selection probabilities.

    constexpr int kNumCosts = 5;
    float costs[kNumCosts] = {0};
    const float depths[kNumCosts] = {
        curr_param_state.depth, prev_param_state.depth, rand_param_state.depth,
        curr_param_state.depth, rand_param_state.depth};
    const float* normals[kNumCosts] = {
        curr_param_state.normal, prev_param_state.normal,
        rand_param_state.normal, rand_param_state.normal,
        curr_param_state.normal};

    for (int sample = 0; sample < options.num_samples; ++sample) {
      const float rand_prob = hiprand_uniform(&rand_state) - FLT_EPSILON;

      pcc_computer.src_image_idx = -1;
      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        const float prob = sampling_probs[image_idx];
        if (prob > rand_prob) {
          pcc_computer.src_image_idx = image_idx;
          break;
        }
      }

      if (pcc_computer.src_image_idx == -1) {
        continue;
      }

      costs[0] += cost_map.Get(row, col, pcc_computer.src_image_idx);
      if (kGeomConsistencyTerm) {
        costs[0] += options.geom_consistency_regularizer *
                    ComputeGeomConsistencyCost(
                        row, col, depths[0], pcc_computer.src_image_idx,
                        options.geom_consistency_max_cost);
      }

      for (int i = 1; i < kNumCosts; ++i) {
        pcc_computer.depth = depths[i];
        pcc_computer.normal = normals[i];
        costs[i] += pcc_computer.Compute();
        if (kGeomConsistencyTerm) {
          costs[i] += options.geom_consistency_regularizer *
                      ComputeGeomConsistencyCost(
                          row, col, depths[i], pcc_computer.src_image_idx,
                          options.geom_consistency_max_cost);
        }
      }
    }

    // Find the parameters of the minimum cost.
    const int min_cost_idx = FindMinCost<kNumCosts>(costs);
    const float best_depth = depths[min_cost_idx];
    const float* best_normal = normals[min_cost_idx];

    // Save best new parameters.
    depth_map.Set(row, col, best_depth);
    normal_map.SetSlice(row, col, best_normal);

    // Use the new cost to recompute the updated forward message and
    // the selection probability.
    pcc_computer.depth = best_depth;
    pcc_computer.normal = best_normal;
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Determine the cost for best depth.
      float cost;
      if (min_cost_idx == 0) {
        cost = cost_map.Get(row, col, image_idx);
      } else {
        pcc_computer.src_image_idx = image_idx;
        cost = pcc_computer.Compute();
        cost_map.Set(row, col, image_idx, cost);
      }

      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);
      forward_message[image_idx] = alpha;
      sel_prob_map.Set(row, col, image_idx, prob);
    }

    if (kFilterPhotoConsistency || kFilterGeomConsistency) {
      int num_consistent = 0;

      float best_point[3];
      ComputePointAtDepth(row, col, best_depth, best_point);

      const float min_ncc_prob =
          likelihood_computer.ComputeNCCProb(1.0f - options.filter_min_ncc);
      const float cos_min_triangulation_angle =
          cos(options.filter_min_triangulation_angle);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        float cos_triangulation_angle;
        float cos_incident_angle;
        ComputeViewingAngles(best_point, best_normal, image_idx,
                             &cos_triangulation_angle, &cos_incident_angle);
        if (cos_triangulation_angle > cos_min_triangulation_angle ||
            cos_incident_angle <= 0.0f) {
          continue;
        }

        if (!kFilterGeomConsistency) {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else if (!kFilterPhotoConsistency) {
          if (ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
              options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob &&
              ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
                  options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        }
      }

      if (num_consistent < options.filter_min_num_consistent) {
        depth_map.Set(row, col, 0.0f);
        normal_map.Set(row, col, 0, 0.0f);
        normal_map.Set(row, col, 1, 0.0f);
        normal_map.Set(row, col, 2, 0.0f);
        for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
          consistency_mask.Set(row, col, image_idx, 0);
        }
      }
    }

    // Update previous depth for next row.
    prev_param_state.depth = best_depth;
    for (int i = 0; i < 3; ++i) {
      prev_param_state.normal[i] = best_normal[i];
    }
  }

  if (col < cost_map.GetWidth()) {
    rand_state_map.Set(0, col, rand_state);
  }
}

PatchMatchCuda::PatchMatchCuda(const PatchMatchOptions& options,
                               const PatchMatch::Problem& problem)
    : options_(options),
      problem_(problem),
      ref_width_(0),
      ref_height_(0),
      rotation_in_half_pi_(0) {
  SetBestCudaDevice(std::stoi(options_.gpu_index));
  InitRefImage();
  InitSourceImages();
  InitTransforms();
  InitWorkspaceMemory();
}

PatchMatchCuda::~PatchMatchCuda() {
  for (size_t i = 0; i < 4; ++i) {
    poses_device_[i].reset();
  }
}

void PatchMatchCuda::Run() {
#define CASE_WINDOW_RADIUS(window_radius, window_step)                        \
  case window_radius:                                                         \
    if (options_.pm_algo == "COLMAP"){                                        \
      std::cout << "Using COLMAP Algo." << std::endl;                         \
      RunWithWindowSizeAndStep<2 * window_radius + 1, window_step>();         \
    }                                                                         \
    else if (options_.pm_algo == "ACMM"){                                     \
      std::cout << "Using ACMM Algo." << std::endl;                           \
      ACMMRunWithWindowSizeAndStep<2 * window_radius + 1, window_step>();     \
    }                                                                         \
    else std::cerr << "Error: Algo not supported" << std::endl;               \
    break;

#define CASE_WINDOW_STEP(window_step)                                 \
  case window_step:                                                   \
    switch (options_.window_radius) {                                 \
      CASE_WINDOW_RADIUS(1, window_step)                              \
      CASE_WINDOW_RADIUS(2, window_step)                              \
      CASE_WINDOW_RADIUS(3, window_step)                              \
      CASE_WINDOW_RADIUS(4, window_step)                              \
      CASE_WINDOW_RADIUS(5, window_step)                              \
      CASE_WINDOW_RADIUS(6, window_step)                              \
      CASE_WINDOW_RADIUS(7, window_step)                              \
      CASE_WINDOW_RADIUS(8, window_step)                              \
      CASE_WINDOW_RADIUS(9, window_step)                              \
      CASE_WINDOW_RADIUS(10, window_step)                             \
      CASE_WINDOW_RADIUS(11, window_step)                             \
      CASE_WINDOW_RADIUS(12, window_step)                             \
      CASE_WINDOW_RADIUS(13, window_step)                             \
      CASE_WINDOW_RADIUS(14, window_step)                             \
      CASE_WINDOW_RADIUS(15, window_step)                             \
      CASE_WINDOW_RADIUS(16, window_step)                             \
      CASE_WINDOW_RADIUS(17, window_step)                             \
      CASE_WINDOW_RADIUS(18, window_step)                             \
      CASE_WINDOW_RADIUS(19, window_step)                             \
      CASE_WINDOW_RADIUS(20, window_step)                             \
      default: {                                                      \
        std::cerr << "Error: Window size not supported" << std::endl; \
        break;                                                        \
      }                                                               \
    }                                                                 \
    break;

  switch (options_.window_step) {
    CASE_WINDOW_STEP(1)
    CASE_WINDOW_STEP(2)
    default: {
      std::cerr << "Error: Window step not supported" << std::endl;
      break;
    }
  }

#undef SWITCH_WINDOW_RADIUS
#undef CALL_RUN_FUNC
}

DepthMap PatchMatchCuda::GetDepthMap() const {
  return DepthMap(depth_map_->CopyToMat(), options_.depth_min,
                  options_.depth_max);
}

NormalMap PatchMatchCuda::GetNormalMap() const {
  return NormalMap(normal_map_->CopyToMat());
}

Mat<float> PatchMatchCuda::GetSelProbMap() const {
  return prev_sel_prob_map_->CopyToMat();
}

std::vector<int> PatchMatchCuda::GetConsistentImageIdxs() const {
  const Mat<uint8_t> mask = consistency_mask_->CopyToMat();
  std::vector<int> consistent_image_idxs;
  std::vector<int> pixel_consistent_image_idxs;
  pixel_consistent_image_idxs.reserve(mask.GetDepth());
  for (size_t r = 0; r < mask.GetHeight(); ++r) {
    for (size_t c = 0; c < mask.GetWidth(); ++c) {
      pixel_consistent_image_idxs.clear();
      for (size_t d = 0; d < mask.GetDepth(); ++d) {
        if (mask.Get(r, c, d)) {
          pixel_consistent_image_idxs.push_back(problem_.src_image_idxs[d]);
        }
      }
      if (pixel_consistent_image_idxs.size() > 0) {
        consistent_image_idxs.push_back(c);
        consistent_image_idxs.push_back(r);
        consistent_image_idxs.push_back(pixel_consistent_image_idxs.size());
        consistent_image_idxs.insert(consistent_image_idxs.end(),
                                     pixel_consistent_image_idxs.begin(),
                                     pixel_consistent_image_idxs.end());
      }
    }
  }
  return consistent_image_idxs;
}

template <int kWindowSize, int kWindowStep>
void PatchMatchCuda::RunWithWindowSizeAndStep() {
  // Wait for all initializations to finish.
  CUDA_SYNC_AND_CHECK();

  CudaTimer total_timer;
  CudaTimer init_timer;

  ComputeCudaConfig();
  ComputeInitialCost<kWindowSize, kWindowStep>
      <<<sweep_grid_size_, sweep_block_size_>>>(
          *cost_map_, *depth_map_, *normal_map_, *ref_image_->sum_image,
          *ref_image_->squared_sum_image, options_.sigma_spatial,
          options_.sigma_color);
  CUDA_SYNC_AND_CHECK();

  init_timer.Print("Initialization");

  const float total_num_steps = options_.num_iterations * 4;

  SweepOptions sweep_options;
  sweep_options.depth_min = options_.depth_min;
  sweep_options.depth_max = options_.depth_max;
  sweep_options.sigma_spatial = options_.sigma_spatial;
  sweep_options.sigma_color = options_.sigma_color;
  sweep_options.num_samples = options_.num_samples;
  sweep_options.ncc_sigma = options_.ncc_sigma;
  sweep_options.min_triangulation_angle =
      DEG2RAD(options_.min_triangulation_angle);
  sweep_options.incident_angle_sigma = options_.incident_angle_sigma;
  sweep_options.geom_consistency_regularizer =
      options_.geom_consistency_regularizer;
  sweep_options.geom_consistency_max_cost = options_.geom_consistency_max_cost;
  sweep_options.filter_min_ncc = options_.filter_min_ncc;
  sweep_options.filter_min_triangulation_angle =
      DEG2RAD(options_.filter_min_triangulation_angle);
  sweep_options.filter_min_num_consistent = options_.filter_min_num_consistent;
  sweep_options.filter_geom_consistency_max_cost =
      options_.filter_geom_consistency_max_cost;

  for (int iter = 0; iter < options_.num_iterations; ++iter) {
    CudaTimer iter_timer;

    for (int sweep = 0; sweep < 4; ++sweep) {
      CudaTimer sweep_timer;

      // Expenentially reduce amount of perturbation during the optimization.
      sweep_options.perturbation = 1.0f / std::pow(2.0f, iter + sweep / 4.0f);

      // Linearly increase the influence of previous selection probabilities.
      sweep_options.prev_sel_prob_weight =
          static_cast<float>(iter * 4 + sweep) / total_num_steps;

      const bool last_sweep = iter == options_.num_iterations - 1 && sweep == 3;

#define CALL_SWEEP_FUNC                                                  \
  SweepFromTopToBottom<kWindowSize, kWindowStep, kGeomConsistencyTerm,   \
                       kFilterPhotoConsistency, kFilterGeomConsistency>  \
      <<<sweep_grid_size_, sweep_block_size_>>>(                         \
          *global_workspace_, *rand_state_map_, *cost_map_, *depth_map_, \
          *normal_map_, *consistency_mask_, *sel_prob_map_,              \
          *prev_sel_prob_map_, *ref_image_->sum_image,                   \
          *ref_image_->squared_sum_image, sweep_options);

      if (last_sweep) {
        if (options_.filter) {
          consistency_mask_.reset(new GpuMat<uint8_t>(cost_map_->GetWidth(),
                                                      cost_map_->GetHeight(),
                                                      cost_map_->GetDepth()));
          consistency_mask_->FillWithScalar(0);
        }
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = true;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        } else {
          const bool kGeomConsistencyTerm = false;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        }
      } else {
        const bool kFilterPhotoConsistency = false;
        const bool kFilterGeomConsistency = false;
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          CALL_SWEEP_FUNC
        } else {
          const bool kGeomConsistencyTerm = false;
          CALL_SWEEP_FUNC
        }
      }

#undef CALL_SWEEP_FUNC

      CUDA_SYNC_AND_CHECK();

      Rotate();

      // Rotate selected image map.
      if (last_sweep && options_.filter) {
        std::unique_ptr<GpuMat<uint8_t>> rot_consistency_mask_(
            new GpuMat<uint8_t>(cost_map_->GetWidth(), cost_map_->GetHeight(),
                                cost_map_->GetDepth()));
        consistency_mask_->Rotate(rot_consistency_mask_.get());
        consistency_mask_.swap(rot_consistency_mask_);
      }

      sweep_timer.Print(" Sweep " + std::to_string(sweep + 1));
    }

    iter_timer.Print("Iteration " + std::to_string(iter + 1));
  }

  total_timer.Print("Total");
}

////////////////////////////////////////////////////////////////////////////////////////   ACMM ///////////////////////////////////////////////////////////////////////////////////////////
template <int kWindowSize>
__device__ inline void ACMMReadRefImageIntoSharedMemory(float* local_image,
                                                          const int row,
                                                          const int col,
                                                          const int thread_idx_x,
                                                          const int thread_idx_y) {
  // For the first row, read the entire block into shared memory. For all
  // consecutive rows, it is only necessary to shift the rows in shared memory
  // up by one element and then read in a new row at the bottom of the shared
  // memory. Note that this assumes that the calling loop starts with the first
  // row and then consecutively reads in a new row.  
  const int kWindowRadius = kWindowSize / 2;
  const int base_start_idx = kWindowRadius * (THREADS_PER_BLOCK + 2 * kWindowRadius);
  const int cur_idx = base_start_idx + thread_idx_y * (THREADS_PER_BLOCK + 2 * kWindowRadius) + kWindowRadius + thread_idx_x;
  
  //read the left data exceed the local image
  if(thread_idx_x == 0){
    for(int i = 1; i <= kWindowRadius; i++){
      local_image[cur_idx - i] = tex2D(ref_image_texture, col - i, row);
    }
  }

  //read the right data exceed the local image
  if(thread_idx_x == THREADS_PER_BLOCK - 1){
    for(int i = 1; i <= kWindowRadius; i++){
      local_image[cur_idx + i] = tex2D(ref_image_texture, col + i, row);
    }
  }

  //read the up data exceed the local image
  if(thread_idx_y == 0){
    for(int i = 1; i <= kWindowRadius; i++){
      local_image[cur_idx - i * (THREADS_PER_BLOCK + 2 * kWindowRadius)] = tex2D(ref_image_texture, col, row - i);
    }
  }

  //read the down data exceed the local image
  if(thread_idx_y == THREADS_PER_BLOCK - 1){
    for(int i = 1; i <= kWindowRadius; i++){
      local_image[cur_idx + i * (THREADS_PER_BLOCK + 2 * kWindowRadius)] = tex2D(ref_image_texture, col, row + i);
    }
  }

  //read the left up data exceed the local image
  if(thread_idx_x == 0 && thread_idx_y == 0){
    for(int i = 1; i <= kWindowRadius; i++){
      for(int j = 1; j <= kWindowRadius; j++)
        local_image[cur_idx - i * (THREADS_PER_BLOCK + 2 * kWindowRadius) - j] = tex2D(ref_image_texture, col - j, row - i);
    }
  }

  //read the left down data exceed the local image
  if(thread_idx_x == 0 && thread_idx_y == THREADS_PER_BLOCK - 1){
    for(int i = 1; i <= kWindowRadius; i++){
      for(int j = 1; j <= kWindowRadius; j++)
        local_image[cur_idx + i * (THREADS_PER_BLOCK + 2 * kWindowRadius) - j] = tex2D(ref_image_texture, col - j, row + i);
    }
  }

  //read the right up data exceed the local image
  if(thread_idx_x == THREADS_PER_BLOCK - 1 && thread_idx_y == 0){
    for(int i = 1; i <= kWindowRadius; i++){
      for(int j = 1; j <= kWindowRadius; j++)
        local_image[cur_idx - i * (THREADS_PER_BLOCK + 2 * kWindowRadius) + j] = tex2D(ref_image_texture, col + j, row - i);
    }
  }

  //read the right down data exceed the local image
  if(thread_idx_x == THREADS_PER_BLOCK - 1 && thread_idx_y == THREADS_PER_BLOCK - 1){
    for(int i = 1; i <= kWindowRadius; i++){
      for(int j = 1; j <= kWindowRadius; j++)
        local_image[cur_idx + i * (THREADS_PER_BLOCK + 2 * kWindowRadius) + j] = tex2D(ref_image_texture, col + j, row + i);
    }
  }

  // this pixel
  local_image[cur_idx] = tex2D(ref_image_texture, col, row);

  //sync
  __syncthreads();
}

template <int kWindowSize, int kWindowStep>
__global__ void ACMMComputeInitialCost(GpuMat<float> cost_map,
                                        GpuMat<float> depth_map,
                                        const GpuMat<float> normal_map,
                                        const GpuMat<float> ref_sum_image,
                                        const GpuMat<float> ref_squared_sum_image,
                                        const float sigma_spatial,
                                        const float sigma_color) {
    const int thread_Idx_x = threadIdx.x;
    const int thread_Idx_y = threadIdx.y;

    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
  
    __shared__ float local_ref_image[(THREADS_PER_BLOCK + 2 * (kWindowSize / 2)) * (THREADS_PER_BLOCK + 2 * (kWindowSize / 2))];

    PhotoConsistencyCostComputer<kWindowSize, kWindowStep> pcc_computer(
      sigma_spatial, sigma_color);
    pcc_computer.local_ref_image = local_ref_image;
    pcc_computer.row = row;
    pcc_computer.col = col;

    float normal[3];
    pcc_computer.normal = normal;


    // // Note that this must be executed even for pixels outside the borders,
    // // since pixels are used in the local neighborhood of the current pixel.

    ACMMReadRefImageIntoSharedMemory<kWindowSize>(local_ref_image, row, col, thread_Idx_x, thread_Idx_y);

    if (col < cost_map.GetWidth() && row < cost_map.GetHeight()) {
        pcc_computer.depth = depth_map.Get(row, col);
        normal_map.GetSlice(row, col, normal);
        
        pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
        pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

        for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
          pcc_computer.src_image_idx = image_idx;
          cost_map.Set(row, col, image_idx, pcc_computer.ACMMCompute_shared(thread_Idx_x, thread_Idx_y));
        }
    }
}

template <typename T>
__device__ inline T Get_cu(T* array_ptr_, const size_t row, const size_t col,
                            const size_t slice, const size_t height_, const size_t pitch_) {
  return *((T*)((char*)array_ptr_ + pitch_ * (slice * height_ + row)) + col);
}

template <typename T>
__device__ inline void Set_cu(T* array_ptr_, const size_t row, const size_t col,
                            const size_t slice, const size_t height_, const size_t pitch_, T value) {
   *((T*)((char*)array_ptr_ + pitch_ * (slice * height_ + row)) + col) = value;
}

__device__ inline float AvgCost(float* cost_map, const size_t cost_map_pitch, const size_t cost_map_height, const size_t cost_map_depth, int row, int col){
  float cur_cost = 0.0f;
  //printf("%d\n", cost_map_depth);
  for(size_t image_idx = 0; image_idx < cost_map_depth; image_idx++){
    cur_cost += Get_cu<float>(cost_map, row, col, image_idx, cost_map_height, cost_map_pitch);
    //cur_cost += image_idx;
  }
  cur_cost /= cost_map_depth;
  return cur_cost;
}

__device__ inline void adjustFloatMaxHeap(float * minCost, int * pt_index, int root) {
  while (root < 8) {
    int lch = 2 * root + 1;
    int rch = lch + 1;
    int index = root;

    if (rch < 8 && (minCost[rch] > minCost[index]) ) {
      index = rch;
    }
    if (lch < 8 && (minCost[lch] > minCost[index]) ) {
      index = lch;
    }

    if (index != root) {

      float tmp = minCost[index];
      minCost[index] = minCost[root];
      minCost[root] = tmp;

      int pt_row = pt_index[2 * index];
      int pt_col = pt_index[2 * index + 1];
      pt_index[2 * index] = pt_index[2 * root];
      pt_index[2 * index + 1] = pt_index[2 * root + 1];
      pt_index[2 * root] = pt_row;
      pt_index[2 * root + 1] = pt_col;

      root = index;
    }
    else {
      break;
    }
  }
}

__device__ inline void CheckBoardSampler(float* cost_map,
                                          size_t cost_map_pitch, 
                                          int row, int col,
                                          const size_t rows, const size_t cols, int neighbors,
                                          int upVstep, int downVstep, int leftVstep, int rightVstep,
                                          int upStripStep, int downStripStep, int leftStripStep, int rightStripStep,
                                          float* minCost,
                                          int* pt_index){
  {
    //UP
    //Up V sample
    if (row - 1 >= 0) {
      float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - 1, col);
      if (cost < minCost[0]) {
        minCost[0] = cost;
        pt_index[0] = row - 1;
        pt_index[1] = col;
        adjustFloatMaxHeap(minCost, pt_index, 0);
      }
    }
    for (int i = 1; i <= upVstep; i++) {
      if (row - 1 - i >= 0) {
        if (col - i >= 0) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - 1 - i, col - i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row - 1 - i;
            pt_index[1] = col - i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
        if (col + i < cols) {
           float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - 1 - i, col + i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row - 1 - i;
            pt_index[1] = col + i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
      }
      else {
        break;
      }
    }
    //Up Strip Sample
    for (int i = 0; i < upStripStep; i++) {
      if (row - 3 - 2 * i >= 0) {
        float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - 3 - 2 * i, col);
        if (cost < minCost[0]) {
          minCost[0] = cost;
          pt_index[0] = row - 3 - 2 * i;
          pt_index[1] = col;
          adjustFloatMaxHeap(minCost, pt_index, 0);
        }
      }
      else {
        break;
      }
    }
  }
  {
    //DOWN
    if (row + 1 < rows) {
      float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + 1, col);
      if (cost < minCost[0]) {
        minCost[0] = cost;
        pt_index[0] = row + 1;
        pt_index[1] = col;
        adjustFloatMaxHeap(minCost, pt_index, 0);
      }
    }
    for (int i = 1; i <= downVstep; i++) {
      if (row + 1 + i < rows) {
        if (col - i >= 0) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + 1 + i, col - i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row + 1 + i;
            pt_index[1] = col - i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
        if (col + i < cols) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + 1 + i, col + i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row + 1 + i;
            pt_index[1] = col + i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
      }
      else {
        break;
      }
    }
    //Down Strip sample
    for (int i = 0; i < downStripStep; i++) {
      if (row + 3 + 2 * i < rows) {
        float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + 3 + 2 * i, col);
        if (cost < minCost[0]) {
          minCost[0] = cost;
          pt_index[0] = row + 3 + 2 * i;
          pt_index[1] = col;
          adjustFloatMaxHeap(minCost, pt_index, 0);
        }
      }
      else {
        break;
      }
    }
  }
  {
    //LEFT
    //Left V sample
    if (col - 1 >= 0) {
      float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row, col - 1);
      if (cost < minCost[0] ) {
        minCost[0] = cost;
        pt_index[0] = row;
        pt_index[1] = col - 1;
        adjustFloatMaxHeap(minCost, pt_index, 0);
      }
    }
    for (int i = 1; i <= leftVstep; i++) {
      if (col - 1 - i >= 0) {
        if (row - i >= 0) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - i, col - 1 - i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row - i;
            pt_index[1] = col - 1 - i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
        if (row + i < rows) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + i, col - 1 - i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row + i;
            pt_index[1] = col - 1 - i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
      }
      else {
        break;
      }
    }
    //Left Strip sample
    for (int i = 0; i < leftStripStep; i++) {
      if (col - 3 - 2 * i >= 0) {
        float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row, col - 3 - 2 * i);
        if (cost < minCost[0]) {
          minCost[0] = cost;
          pt_index[0] = row;
          pt_index[1] = col - 3 - 2 * i;
          adjustFloatMaxHeap(minCost, pt_index, 0);
        }
      }
      else {
        break;
      }
    }
  }
  {
    //RIGHT
    //Right V sample
    if (col + 1 < cols) {
      float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row, col + 1);
      if (cost < minCost[0]) {
        minCost[0] = cost;
        pt_index[0] = row;
        pt_index[1] = col + 1;
        adjustFloatMaxHeap(minCost, pt_index, 0);
      }
    }
    for (int i = 1; i <= rightVstep; i++) {
      if (col + 1 + i < cols) {
        if (row - i >= 0) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row - i, col + 1 + i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row - i;
            pt_index[1] = col + 1 + i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
        if (row + i < rows) {
          float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row + i, col + 1 + i);
          if (cost < minCost[0]) {
            minCost[0] = cost;
            pt_index[0] = row + i;
            pt_index[1] = col + 1 + i;
            adjustFloatMaxHeap(minCost, pt_index, 0);
          }
        }
      }
      else {
        break;
      }
    }
    //Right Strip sample
    for (int i = 0; i < rightStripStep; i++) {
      if (col + 3 + 2 * i < cols) {
        float cost = AvgCost(cost_map, cost_map_pitch, rows, neighbors, row, col + 3 + 2 * i);
        if (cost < minCost[0]) {
          minCost[0] = cost;
          pt_index[0] = row;
          pt_index[1] = col + 3 + 2 * i;
          adjustFloatMaxHeap(minCost, pt_index, 0);
        }
      }
      else {
        break;
      }
    }
  }
}

template<int kWindowSize, int kWindowStep, bool kGeomConsistencyTerm = false>
__global__ void ACMMCheckerBoard_cu(GpuMat<float> cost_map,
                                          GpuMat<float> depth_map,
                                          GpuMat<float> normal_map,
                                          GpuMat<float> M_map,
                                          GpuMat<int> last_important_view_map,
                                          GpuMat<float> view_weight_map,
                                          const GpuMat<float> ref_sum_image,
                                          const GpuMat<float> ref_squared_sum_image,
                                          GpuMat<hiprandState> rand_state_map,
                                          int iter,
                                          float sigma_spatial,
                                          float sigma_color,
                                          float depth_min,
                                          float depth_max
                                          bool isBlack,
                                          float geom_lamda = 0.0f) {

    const int thread_Idx_x = threadIdx.x;
    const int thread_Idx_y = threadIdx.y;

    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    
    __shared__ float local_ref_image[(THREADS_PER_BLOCK + 2 * (kWindowSize / 2)) * (THREADS_PER_BLOCK + 2 * (kWindowSize / 2))];

    PhotoConsistencyCostComputer<kWindowSize, kWindowStep> pcc_computer(
      sigma_spatial, sigma_color);
    pcc_computer.local_ref_image = local_ref_image;
    pcc_computer.row = row;
    pcc_computer.col = col;

    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    
    ACMMReadRefImageIntoSharedMemory<kWindowSize>(local_ref_image, row, col, thread_Idx_x, thread_Idx_y);
    
  
    if(isBlack && ((threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0) || (threadIdx.x % 2 != 0 && threadIdx.y % 2 != 0)) ||
        !isBlack && ((threadIdx.x % 2 != 0 && threadIdx.y % 2 == 0) || (threadIdx.x % 2 == 0 && threadIdx.y % 2 != 0))){
      // process black pixel
      if (col < cost_map.GetWidth() && row < cost_map.GetHeight()) {
          pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
          pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

          // 8 hypo and correspond uv
          float minCost[8] = {pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost, pcc_computer.kMaxCost};
          int pt_index[16] = {row,col, row,col, row,col, row,col, row,col, row,col, row,col, row,col};
          
          // i. select the 8 hypo with min cost
          CheckBoardSampler(cost_map.GetPtr(), cost_map.GetPitch(), row, col, cost_map.GetHeight(), cost_map.GetWidth(), cost_map.GetDepth(), 
                            V_step_.Get(row, col, 0), V_step_.Get(row, col, 1), V_step_.Get(row, col, 2), V_step_.Get(row, col, 3),
                            S_step_.Get(row, col, 0), S_step_.Get(row, col, 1), S_step_.Get(row, col, 2), S_step_.Get(row, col, 3),
                            minCost, pt_index);

          //update the search area
          



          // 9 hypo: 8 selected and the current
          float normals_0[3];
          float normals_1[3];
          float normals_2[3];
          float normals_3[3];
          float normals_4[3];
          float normals_5[3];
          float normals_6[3];
          float normals_7[3];
          float normals_8[3];
          normal_map.GetSlice(pt_index[2 * 0], pt_index[2 * 0 + 1], normals_0);
          normal_map.GetSlice(pt_index[2 * 1], pt_index[2 * 1 + 1], normals_1);
          normal_map.GetSlice(pt_index[2 * 2], pt_index[2 * 2 + 1], normals_2);
          normal_map.GetSlice(pt_index[2 * 3], pt_index[2 * 3 + 1], normals_3);
          normal_map.GetSlice(pt_index[2 * 4], pt_index[2 * 4 + 1], normals_4);
          normal_map.GetSlice(pt_index[2 * 5], pt_index[2 * 5 + 1], normals_5);
          normal_map.GetSlice(pt_index[2 * 6], pt_index[2 * 6 + 1], normals_6);
          normal_map.GetSlice(pt_index[2 * 7], pt_index[2 * 7 + 1], normals_7);
          normal_map.GetSlice(row, col, normals_8);	
          const float* normals[9] = {normals_0, normals_1, normals_2, normals_3, normals_4, normals_5, normals_6, normals_7, normals_8};
          
          // N: neighbor view
          size_t N = cost_map.GetDepth();

          /*
          * ii. Compute M Matrix
          */
          for (size_t i = 0; i < 9; i++) {
            pcc_computer.normal = normals[i];
            if(i == 8){
              pcc_computer.depth = depth_map.Get(row, col);
            }
            else{
              pcc_computer.depth = depth_map.Get(pt_index[2 * i], pt_index[2 * i + 1]);
            }
          
            for (size_t image_idx = 0; image_idx < N; image_idx++) {
                  pcc_computer.src_image_idx = image_idx;
                  float c = pcc_computer.ACMMCompute_shared(thread_Idx_x, thread_Idx_y);
                  M_map.Set(row, col, N * i + image_idx, c);
            }
          }

          /*
          * iii. Computer viewWeight
          */
          float init_good_threshold = 0.8f;
          float bad_threshold = 1.2f;
          int viewWeight_n1 = 2;
          int viewWeight_n2 = 3;
          float viewWeight_alpha = 90.0f;
          float viewWeight_belta = 0.3f;

          float good_threshold =init_good_threshold * exp(-iter * iter / viewWeight_alpha);
          float maxWeight = 0.0f;
          int lastImportant = -1;

          for (size_t image_idx = 0; image_idx < N; image_idx++) {
            float weight = 0.0f;
            int S_good_size = 0;
            float S_good_score = 0.0f;
            int S_bad_size = 0;

            for (size_t i = 0; i < 9; i++) {
              float mij = M_map.Get(row, col, N * i + image_idx);
              //formula (4)
              if (mij < good_threshold) {
                S_good_score += exp(-mij * mij / (2 * viewWeight_belta * viewWeight_belta));
                S_good_size += 1;
              }
              if (mij > bad_threshold) {
                S_bad_size += 1;
              }
            }
      
            int I = 1;
            if (image_idx == last_important_view_map.Get(row, col)) {
              I = 1;
            }
            else {
              I = 0;
            }
            
            //formula (5)
            if (S_good_size > viewWeight_n1 && S_bad_size < viewWeight_n2) {
              S_good_score = S_good_score / S_good_size;
              weight = (I + 1) * S_good_score;
            }
            else {
              weight = 0.2 * (I);
            }
            // update the lastImportant if this is more important view
            if (weight > maxWeight) {
              lastImportant = image_idx;
              maxWeight = weight;
            }
            view_weight_map.Set(row, col, image_idx, weight);
          }
          last_important_view_map.Set(row, col, lastImportant);

          // select the hypo with min cost
          float minScore = pcc_computer.kMaxCost;
          int minHypo = 8;
          float e_depth = 0.0f;
          for (int i = 0; i < 9; i++) {
            float score = 0.0f;
            float weight_sum = 0.0f;
            
            if(kGeomConsistencyTerm){
              // if use geom, get each hypo depth to reproject
              if(i != 8){
                e_depth = depth_map.Get(pt_index[2 * i], pt_index[2 * i + 1]);
              }
              else{
                e_depth = depth_map.Get(row, col);
              }
            }
            for (size_t image_idx = 0; image_idx < N; image_idx++) {
              float mij = M_map.Get(row, col, N * i + image_idx);
              float eij = 0.0f;
              if(kGeomConsistencyTerm){
                // if use geom, compute reproject error
                eij = ComputeGeomConsistencyCost(row, col, e_depth, image_idx, 2.0f);
              }
              float viewWeight = view_weight_map.Get(row, col, image_idx);
              if (mij < pcc_computer.kMaxCost) {
                score += (mij + geom_lamda * eij) * viewWeight;
                weight_sum += viewWeight;
              }
            }

            if (weight_sum != 0.0f) {
              score = score / weight_sum;
              if (score < 0 || score > pcc_computer.kMaxCost) {
                score = pcc_computer.kMaxCost;
              }
              if (score < minScore) {
                minScore = score;
                minHypo = i;
              }
            }
          }
          
          // update the current state
          const float *best_normal = normals[minHypo];
          float best_depth = depth_map.Get(row, col);
          if(minHypo != 8){
            best_depth = depth_map.Get(pt_index[2 * minHypo], pt_index[2 * minHypo + 1]);
          }
          depth_map.Set(row, col, best_depth);
          normal_map.SetSlice(row, col, best_normal);
          for (size_t image_idx = 0; image_idx < N; image_idx++) {
            cost_map.Set(row, col, image_idx, M_map.Get(row, col, N * minHypo + image_idx));
          }
      }
  }
  
  __syncthreads();
}

template<int kWindowSize, int kWindowStep>
__global__ void RefineMent(GpuMat<float> cost_map,
                      GpuMat<float> depth_map,
                                      GpuMat<float> normal_map,
                                      GpuMat<float> M_map,
                                      GpuMat<int> last_important_view_map,
                                      GpuMat<float> view_weight_map,
                                      const GpuMat<float> ref_sum_image,
                                         const GpuMat<float> ref_squared_sum_image,
                                         GpuMat<hiprandState> rand_state_map,
                      int iter,
                      float sigma_spatial,
                      float sigma_color,
                      float depth_min,
                      float depth_max) {

    const int thread_Idx_x = threadIdx.x;
    const int thread_Idx_y = threadIdx.y;

    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    
    //__shared__ float local_ref_image[9 * THREADS_PER_BLOCK * THREADS_PER_BLOCK];
    __shared__ float local_ref_image[(THREADS_PER_BLOCK + 2 * (kWindowSize / 2)) * (THREADS_PER_BLOCK + 2 * (kWindowSize / 2))];

    PhotoConsistencyCostComputer<kWindowSize, kWindowStep> pcc_computer(
      sigma_spatial, sigma_color);
    pcc_computer.local_ref_image = local_ref_image;
    pcc_computer.row = row;
    pcc_computer.col = col;

    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    
    //                                          
    ACMMReadRefImageIntoSharedMemory<kWindowSize>(local_ref_image, row, col,thread_Idx_x, thread_Idx_y);

  if (col < cost_map.GetWidth() && row < cost_map.GetHeight()) {	
    //depth_map.Set(row, col, local_ref_image[(thread_Idx_y + kWindowRadius)* (THREADS_PER_BLOCK + 2 * kWindowRadius) + kWindowRadius + thread_Idx_x]);
      pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
      pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

      float perturbation = 1.0f / pow(2.0f, iter);
      hiprandState rand_state = rand_state_map.Get(row, col);

      float cur_normal[3];
      normal_map.GetSlice(row, col, cur_normal);

      float rnd_normal[3];
      float prb_normal[3];
      
      GenerateRandomNormal(row, col, &rand_state, rnd_normal);
      PerturbNormal(row, col, perturbation * M_PI, cur_normal, &rand_state, prb_normal);

      float cur_depth = depth_map.Get(row, col);
      float rnd_depth;
      float prb_depth;

      rnd_depth = GenerateRandomDepth(depth_min, depth_max, &rand_state);
      prb_depth = PerturbDepth(perturbation, cur_depth, &rand_state);
          

        //0: prb_norm,rnd_depth
       //1: rnd_norm,rnd_depth
       //2: cur_norm,rnd_depth
       //3: prb_norm,prb_depth
       //4: rnd_norm,prb_depth
       //5: cur_norm,prb_depth
       //6: prb_norm,cur_depth
       //7: rnd_norm,cur_depth
      //8: cur_norm,cur_depth	

       const float hypo_depths[9] = {rnd_depth, rnd_depth, rnd_depth, prb_depth, prb_depth, prb_depth, cur_depth, cur_depth, cur_depth};
      const float* normals[9] = {prb_normal, rnd_normal, cur_normal, prb_normal, rnd_normal, cur_normal, prb_normal, rnd_normal, cur_normal};
    
      size_t N = cost_map.GetDepth();
      /*
      * ii. Compute M Matrix
      */
      // float dis_cost = 0.0f;
      // int index = 0;
      // float M[256];
      // float ViewWeight[32];
      for (size_t i = 0; i < 9; i++) {
        pcc_computer.normal = normals[i];
        pcc_computer.depth = hypo_depths[i];
        
        for (size_t image_idx = 0; image_idx < N; image_idx++) {
              pcc_computer.src_image_idx = image_idx;
              float c = pcc_computer.ACMMCompute_shared(thread_Idx_x, thread_Idx_y);
              M_map.Set(row, col, N * i + image_idx, c);
            }
      }

      // for (int i = 0; i < 9; i++) {
      // 	for (int image_idx = 0; image_idx < N; ++image_idx) {
      // 		dis_cost += M_map.Get(row, col, N * i + image_idx);
      // 	}
      // }
      // dis_cost = dis_cost/(9*N);
      //printf("dis_cost:%f\n", dis_cost);
      //depth_map.Set(row, col, dis_cost);

      /*
      * iii. Computer viewWeight
      */
      float init_good_threshold = 0.8f;
      float bad_threshold = 1.2f;
      int viewWeight_n1 = 2;
      int viewWeight_n2 = 3;
      float viewWeight_alpha = 90.0f;
      float viewWeight_belta = 0.3f;

      float good_threshold =init_good_threshold * exp(-iter * iter / viewWeight_alpha);
      float maxWeight = 0.0f;
      int lastImportant = -1;

      for (size_t image_idx = 0; image_idx < N; image_idx++) {

        float weight = 0.0f;
        int S_good_size = 0;
        float S_good_score = 0.0f;
        int S_bad_size = 0;

        for (size_t i = 0; i < 9; i++) {
          float mij = M_map.Get(row, col, N * i + image_idx);
          //formula (4)
          if (mij < good_threshold) {
            S_good_score += exp(-mij * mij / (2 * viewWeight_belta * viewWeight_belta));
            
            S_good_size += 1;
          }
          if (mij > bad_threshold) {
            S_bad_size += 1;
          }
        }

        int I = 1;
        if (image_idx == last_important_view_map.Get(row, col)) {
          I = 1;
        }
        else {
          I = 0;
        }
        //printf("S_good_size %d\n", S_good_size);
        //printf("S_bad_size %d\n", S_bad_size);
        // //formula (5)
        if (S_good_size > viewWeight_n1 && S_bad_size < viewWeight_n2) {
          S_good_score = S_good_score / S_good_size;
          weight = (I + 1) * S_good_score;
        }
        else {
          weight = 0.2 * (I);
        }
    //printf("mij %f\n", weight);
        if (weight > maxWeight) {
          lastImportant = image_idx;
          maxWeight = weight;
        }
        view_weight_map.Set(row, col, image_idx, weight);
      }
      last_important_view_map.Set(row, col, lastImportant);
      
      // dis_cost = 0;
      // for (int image_idx = 0; image_idx < N; ++image_idx) {
      // 	dis_cost += view_weight_map.Get(row, col, image_idx);
      // }
      
      // dis_cost = dis_cost/(N);
      // printf("dis %f\n", dis_cost);
      // depth_map.Set(row, col, dis_cost);

      float minScore = pcc_computer.kMaxCost;
      int minHypo = 8;

      for (int i = 0; i < 9; i++) {
        float score = 0.0f;
        float weight_sum = 0.0f;
        
        for (size_t image_idx = 0; image_idx < N; image_idx++) {
          float mij = M_map.Get(row, col, N * i + image_idx);
          float viewWeight = view_weight_map.Get(row, col, image_idx);
          if (mij < pcc_computer.kMaxCost) {
            score += mij * viewWeight;
            weight_sum += viewWeight;
          }
        }
        if (weight_sum != 0.0f) {
          score = score / weight_sum;
          if (score < 0 || score > pcc_computer.kMaxCost) {
            score = pcc_computer.kMaxCost;
          }

          if (score < minScore) {
            minScore = score;
            minHypo = i;
          }
        }
      }
      //printf("minHypo %d\n", minHypo);
      const float *best_normal = normals[minHypo];
      float best_depth = hypo_depths[minHypo];
      
      depth_map.Set(row, col, best_depth);
      normal_map.SetSlice(row, col, best_normal);
      for (size_t image_idx = 0; image_idx < N; image_idx++) {
        cost_map.Set(row, col, M_map.Get(row, col, N * minHypo + image_idx));
      }
      rand_state_map.Set(row, col, rand_state);
  }
  __syncthreads();
}

template <int kWindowSize, int kWindowStep>
void PatchMatchCuda::ACMMRunWithWindowSizeAndStep() {
  // Wait for all initializations to finish.
    CUDA_SYNC_AND_CHECK();

    CudaTimer total_timer;
    CudaTimer init_timer;

    ComputeCudaConfig();
    // random init and compute cost
    ACMMComputeInitialCost<kWindowSize, kWindowStep>
      <<<elem_wise_grid_size_, elem_wise_block_size_>>>(
          *cost_map_, *depth_map_, *normal_map_, *ref_image_->sum_image,
          *ref_image_->squared_sum_image, options_.sigma_spatial,
          options_.sigma_color);
    CUDA_SYNC_AND_CHECK();

    init_timer.Print("Initialization");

    bool kGeomConsistencyTerm = false;
    if(options_.){
      kGeomConsistencyTerm = true;
    }
    for(int iter = 0; iter < options_.num_iterations; ++iter) {
      CudaTimer iter_timer;
      CUDA_SYNC_AND_CHECK();
      ACMMCheckerBoard_cu<kWindowSize, kWindowStep, kGeomConsistencyTerm><<<elem_wise_grid_size_, elem_wise_block_size_>>>
      ( *cost_map_, *depth_map_, *normal_map_, *M_map_, *last_important_view_map_, *sel_prob_map_, *ref_image_->sum_image, *ref_image_->squared_sum_image, *rand_state_map_, iter, options_.sigma_spatial,
          options_.sigma_color, options_.depth_min, options_.depth_max, true, options_.geom_consistency_regularizer);
      CUDA_SYNC_AND_CHECK();
      ACMMCheckerBoard_cu<kWindowSize, kWindowStep><<<elem_wise_grid_size_, elem_wise_block_size_>>>
      ( *cost_map_, *depth_map_, *normal_map_, *M_map_, *last_important_view_map_, *sel_prob_map_, *ref_image_->sum_image, *ref_image_->squared_sum_image, *rand_state_map_, iter, options_.sigma_spatial,
          options_.sigma_color, options_.depth_min, options_.depth_max, false);
      CUDA_SYNC_AND_CHECK();
      RefineMent<kWindowSize, kWindowStep><<<elem_wise_grid_size_, elem_wise_block_size_>>>
      ( *cost_map_, *depth_map_, *normal_map_, *M_map_, *last_important_view_map_, *sel_prob_map_, *ref_image_->sum_image, *ref_image_->squared_sum_image, *rand_state_map_, iter, options_.sigma_spatial,
          options_.sigma_color, options_.depth_min, options_.depth_max);
      CUDA_SYNC_AND_CHECK();
      iter_timer.Print("Iteration " + std::to_string(iter + 1));
    }

    total_timer.Print("Total");
}
////////////////////////////////////////////////////////////////////////////////////////   ACMM END ///////////////////////////////////////////////////////////////////////////////////////////


void PatchMatchCuda::ComputeCudaConfig() {
  sweep_block_size_.x = THREADS_PER_BLOCK;
  sweep_block_size_.y = 1;
  sweep_block_size_.z = 1;
  sweep_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  sweep_grid_size_.y = 1;
  sweep_grid_size_.z = 1;

  elem_wise_block_size_.x = THREADS_PER_BLOCK;
  elem_wise_block_size_.y = THREADS_PER_BLOCK;
  elem_wise_block_size_.z = 1;
  elem_wise_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.y =
      (depth_map_->GetHeight() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.z = 1;
}

void PatchMatchCuda::InitRefImage() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  ref_width_ = ref_image.GetWidth();
  ref_height_ = ref_image.GetHeight();

  // Upload to device.
  ref_image_.reset(new GpuMatRefImage(ref_width_, ref_height_));
  const std::vector<uint8_t> ref_image_array =
      ref_image.GetBitmap().ConvertToRowMajorArray();
  ref_image_->Filter(ref_image_array.data(), options_.window_radius,
                     options_.window_step, options_.sigma_spatial,
                     options_.sigma_color);

  ref_image_device_.reset(
      new CudaArrayWrapper<uint8_t>(ref_width_, ref_height_, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);

  // Create texture.
  ref_image_texture.addressMode[0] = hipAddressModeBorder;
  ref_image_texture.addressMode[1] = hipAddressModeBorder;
  ref_image_texture.addressMode[2] = hipAddressModeBorder;
  ref_image_texture.filterMode = hipFilterModePoint;
  ref_image_texture.normalized = false;
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));
}

void PatchMatchCuda::InitSourceImages() {
  // Determine maximum image size.
  size_t max_width = 0;
  size_t max_height = 0;
  for (const auto image_idx : problem_.src_image_idxs) {
    const Image& image = problem_.images->at(image_idx);
    if (image.GetWidth() > max_width) {
      max_width = image.GetWidth();
    }
    if (image.GetHeight() > max_height) {
      max_height = image.GetHeight();
    }
  }

  // Upload source images to device.
  {
    // Copy source images to contiguous memory block.
    const uint8_t kDefaultValue = 0;
    std::vector<uint8_t> src_images_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const Image& image = problem_.images->at(problem_.src_image_idxs[i]);
      const Bitmap& bitmap = image.GetBitmap();
      uint8_t* dest = src_images_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < image.GetHeight(); ++r) {
        memcpy(dest, bitmap.GetScanline(r), image.GetWidth() * sizeof(uint8_t));
        dest += max_width;
      }
    }

    // Upload to device.
    src_images_device_.reset(new CudaArrayWrapper<uint8_t>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_images_device_->CopyToDevice(src_images_host_data.data());

    // Create source images texture.
    src_images_texture.addressMode[0] = hipAddressModeBorder;
    src_images_texture.addressMode[1] = hipAddressModeBorder;
    src_images_texture.addressMode[2] = hipAddressModeBorder;
    src_images_texture.filterMode = hipFilterModeLinear;
    src_images_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_images_texture,
                                          src_images_device_->GetPtr()));
  }

  // Upload source depth maps to device.
  if (options_.geom_consistency) {
    const float kDefaultValue = 0.0f;
    std::vector<float> src_depth_maps_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const DepthMap& depth_map =
          problem_.depth_maps->at(problem_.src_image_idxs[i]);
      float* dest =
          src_depth_maps_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < depth_map.GetHeight(); ++r) {
        memcpy(dest, depth_map.GetPtr() + r * depth_map.GetWidth(),
               depth_map.GetWidth() * sizeof(float));
        dest += max_width;
      }
    }

    src_depth_maps_device_.reset(new CudaArrayWrapper<float>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_depth_maps_device_->CopyToDevice(src_depth_maps_host_data.data());

    // Create source depth maps texture.
    src_depth_maps_texture.addressMode[0] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[1] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[2] = hipAddressModeBorder;
    // TODO: Check if linear interpolation improves results or not.
    src_depth_maps_texture.filterMode = hipFilterModePoint;
    src_depth_maps_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_depth_maps_texture,
                                          src_depth_maps_device_->GetPtr()));
  }
}

void PatchMatchCuda::InitTransforms() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  //////////////////////////////////////////////////////////////////////////////
  // Generate rotated versions (counter-clockwise) of calibration matrix.
  //////////////////////////////////////////////////////////////////////////////

  for (size_t i = 0; i < 4; ++i) {
    ref_K_host_[i][0] = ref_image.GetK()[0];
    ref_K_host_[i][1] = ref_image.GetK()[2];
    ref_K_host_[i][2] = ref_image.GetK()[4];
    ref_K_host_[i][3] = ref_image.GetK()[5];
  }

  // Rotated by 90 degrees.
  std::swap(ref_K_host_[1][0], ref_K_host_[1][2]);
  std::swap(ref_K_host_[1][1], ref_K_host_[1][3]);
  ref_K_host_[1][3] = ref_width_ - 1 - ref_K_host_[1][3];

  // Rotated by 180 degrees.
  ref_K_host_[2][1] = ref_width_ - 1 - ref_K_host_[2][1];
  ref_K_host_[2][3] = ref_height_ - 1 - ref_K_host_[2][3];

  // Rotated by 270 degrees.
  std::swap(ref_K_host_[3][0], ref_K_host_[3][2]);
  std::swap(ref_K_host_[3][1], ref_K_host_[3][3]);
  ref_K_host_[3][1] = ref_height_ - 1 - ref_K_host_[3][1];

  // Extract 1/fx, -cx/fx, fy, -cy/fy.
  for (size_t i = 0; i < 4; ++i) {
    ref_inv_K_host_[i][0] = 1.0f / ref_K_host_[i][0];
    ref_inv_K_host_[i][1] = -ref_K_host_[i][1] / ref_K_host_[i][0];
    ref_inv_K_host_[i][2] = 1.0f / ref_K_host_[i][2];
    ref_inv_K_host_[i][3] = -ref_K_host_[i][3] / ref_K_host_[i][2];
  }

  // Bind 0 degrees version to constant global memory.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[0], sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[0],
                                    sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));

  //////////////////////////////////////////////////////////////////////////////
  // Generate rotated versions of camera poses.
  //////////////////////////////////////////////////////////////////////////////

  float rotated_R[9];
  memcpy(rotated_R, ref_image.GetR(), 9 * sizeof(float));

  float rotated_T[3];
  memcpy(rotated_T, ref_image.GetT(), 3 * sizeof(float));

  // Matrix for 90deg rotation around Z-axis in counter-clockwise direction.
  const float R_z90[9] = {0, 1, 0, -1, 0, 0, 0, 0, 1};

  for (size_t i = 0; i < 4; ++i) {
    const size_t kNumTformParams = 4 + 9 + 3 + 3 + 12 + 12;
    std::vector<float> poses_host_data(kNumTformParams *
                                       problem_.src_image_idxs.size());
    int offset = 0;
    for (const auto image_idx : problem_.src_image_idxs) {
      const Image& image = problem_.images->at(image_idx);

      const float K[4] = {image.GetK()[0], image.GetK()[2], image.GetK()[4],
                          image.GetK()[5]};
      memcpy(poses_host_data.data() + offset, K, 4 * sizeof(float));
      offset += 4;

      float rel_R[9];
      float rel_T[3];
      ComputeRelativePose(rotated_R, rotated_T, image.GetR(), image.GetT(),
                          rel_R, rel_T);
      memcpy(poses_host_data.data() + offset, rel_R, 9 * sizeof(float));
      offset += 9;
      memcpy(poses_host_data.data() + offset, rel_T, 3 * sizeof(float));
      offset += 3;

      float C[3];
      ComputeProjectionCenter(rel_R, rel_T, C);
      memcpy(poses_host_data.data() + offset, C, 3 * sizeof(float));
      offset += 3;

      float P[12];
      ComposeProjectionMatrix(image.GetK(), rel_R, rel_T, P);
      memcpy(poses_host_data.data() + offset, P, 12 * sizeof(float));
      offset += 12;

      float inv_P[12];
      ComposeInverseProjectionMatrix(image.GetK(), rel_R, rel_T, inv_P);
      memcpy(poses_host_data.data() + offset, inv_P, 12 * sizeof(float));
      offset += 12;
    }

    poses_device_[i].reset(new CudaArrayWrapper<float>(
        kNumTformParams, problem_.src_image_idxs.size(), 1));
    poses_device_[i]->CopyToDevice(poses_host_data.data());

    RotatePose(R_z90, rotated_R, rotated_T);
  }

  poses_texture.addressMode[0] = hipAddressModeBorder;
  poses_texture.addressMode[1] = hipAddressModeBorder;
  poses_texture.addressMode[2] = hipAddressModeBorder;
  poses_texture.filterMode = hipFilterModePoint;
  poses_texture.normalized = false;
  CUDA_SAFE_CALL(
      hipBindTextureToArray(poses_texture, poses_device_[0]->GetPtr()));
}

void PatchMatchCuda::InitWorkspaceMemory() {
  rand_state_map_.reset(new GpuMatPRNG(ref_width_, ref_height_));

  depth_map_.reset(new GpuMat<float>(ref_width_, ref_height_));
  if (options_.geom_consistency) {
    const DepthMap& init_depth_map =
        problem_.depth_maps->at(problem_.ref_image_idx);
    depth_map_->CopyToDevice(init_depth_map.GetPtr(),
                             init_depth_map.GetWidth() * sizeof(float));
  } else {
    depth_map_->FillWithRandomNumbers(options_.depth_min, options_.depth_max,
                                      *rand_state_map_);
  }

  normal_map_.reset(new GpuMat<float>(ref_width_, ref_height_, 3));

  // Note that it is not necessary to keep the selection probability map in
  // memory for all pixels. Theoretically, it is possible to incorporate
  // the temporary selection probabilities in the global_workspace_.
  // However, it is useful to keep the probabilities for the entire image
  // in memory, so that it can be exported.
  sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                        problem_.src_image_idxs.size()));
  prev_sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                             problem_.src_image_idxs.size()));
  prev_sel_prob_map_->FillWithScalar(0.5f);

  cost_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                    problem_.src_image_idxs.size()));

  // ACMM
  cost_map_->FillWithScalar(2.0f);

  M_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                    9 * problem_.src_image_idxs.size()));

  M_map_->FillWithScalar(2.0f);

  last_important_view_map_.reset(new GpuMat<int>(ref_width_, ref_height_));

  last_important_view_map_->FillWithScalar(-1);

  view_weight_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                    problem_.src_image_idxs.size()));

  // up down left right
  V_step_.reset(new GpuMat<int>(ref_width_, ref_height_, 4)); 
  V_step_->FillWithScalar(3);
  S_step_.reset(new GpuMat<int>(ref_width_, ref_height_, 4));
  S_step_->FillWithScalar(11);
  
  // ACMM END

  const int ref_max_dim = std::max(ref_width_, ref_height_);
  global_workspace_.reset(
      new GpuMat<float>(ref_max_dim, problem_.src_image_idxs.size(), 2));

  consistency_mask_.reset(new GpuMat<uint8_t>(0, 0, 0));

  ComputeCudaConfig();

  if (options_.geom_consistency) {
    const NormalMap& init_normal_map =
        problem_.normal_maps->at(problem_.ref_image_idx);
    normal_map_->CopyToDevice(init_normal_map.GetPtr(),
                              init_normal_map.GetWidth() * sizeof(float));
  } else {
    InitNormalMap<<<elem_wise_grid_size_, elem_wise_block_size_>>>(
        *normal_map_, *rand_state_map_);
  }
}

void PatchMatchCuda::Rotate() {
  rotation_in_half_pi_ = (rotation_in_half_pi_ + 1) % 4;

  size_t width;
  size_t height;
  if (rotation_in_half_pi_ % 2 == 0) {
    width = ref_width_;
    height = ref_height_;
  } else {
    width = ref_height_;
    height = ref_width_;
  }

  // Rotate random map.
  {
    std::unique_ptr<GpuMatPRNG> rotated_rand_state_map(
        new GpuMatPRNG(width, height));
    rand_state_map_->Rotate(rotated_rand_state_map.get());
    rand_state_map_.swap(rotated_rand_state_map);
  }

  // Rotate depth map.
  {
    std::unique_ptr<GpuMat<float>> rotated_depth_map(
        new GpuMat<float>(width, height));
    depth_map_->Rotate(rotated_depth_map.get());
    depth_map_.swap(rotated_depth_map);
  }

  // Rotate normal map.
  {
    RotateNormalMap<<<elem_wise_grid_size_, elem_wise_block_size_>>>(
        *normal_map_);
    std::unique_ptr<GpuMat<float>> rotated_normal_map(
        new GpuMat<float>(width, height, 3));
    normal_map_->Rotate(rotated_normal_map.get());
    normal_map_.swap(rotated_normal_map);
  }

  // Rotate reference image.
  {
    std::unique_ptr<GpuMatRefImage> rotated_ref_image(
        new GpuMatRefImage(width, height));
    ref_image_->image->Rotate(rotated_ref_image->image.get());
    ref_image_->sum_image->Rotate(rotated_ref_image->sum_image.get());
    ref_image_->squared_sum_image->Rotate(
        rotated_ref_image->squared_sum_image.get());
    ref_image_.swap(rotated_ref_image);
  }

  // Bind rotated reference image to texture.
  ref_image_device_.reset(new CudaArrayWrapper<uint8_t>(width, height, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);
  CUDA_SAFE_CALL(hipUnbindTexture(ref_image_texture));
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));

  // Rotate selection probability map.
  prev_sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
  sel_prob_map_->Rotate(prev_sel_prob_map_.get());
  sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));

  // Rotate cost map.
  {
    std::unique_ptr<GpuMat<float>> rotated_cost_map(
        new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
    cost_map_->Rotate(rotated_cost_map.get());
    cost_map_.swap(rotated_cost_map);
  }

  // Rotate transformations.
  CUDA_SAFE_CALL(hipUnbindTexture(poses_texture));
  CUDA_SAFE_CALL(hipBindTextureToArray(
      poses_texture, poses_device_[rotation_in_half_pi_]->GetPtr()));

  // Rotate calibration.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[rotation_in_half_pi_],
                                    sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[rotation_in_half_pi_],
                         sizeof(float) * 4, 0, hipMemcpyHostToDevice));

  // Recompute Cuda configuration for rotated reference image.
  ComputeCudaConfig();
}

}  // namespace mvs
}  // namespace colmap
